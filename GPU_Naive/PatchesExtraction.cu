#include "hip/hip_runtime.h"
/**
 * PatchesExtraction.cu
 * Contains CUDA kernels and functions for extracting reference, positive,
 * and negative patches from input and output images.
 * This module is crucial for the initial phase of the Patchwise Contrastive Loss computation.
 */

#include "NaiveParallelization.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA_ERROR(call) {                                           \
    hipError_t err = call;                                                \
    if (err != hipSuccess) {                                              \
        fprintf(stderr, "CUDA error at %s %d: %s\n", __FILE__, __LINE__,   \
                hipGetErrorString(err));                                  \
        exit(EXIT_FAILURE);                                                \
    }                                                                      \
}

// ==================== Patches Coordinates ====================

__global__ void generateRandomCoordinatesNaivePara(int* coords, int width, int height, int patchSize, int numPatches, unsigned long long seed) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    if (idx < numPatches) {
        // Generate random x and y coordinates for placing the patches
        coords[2 * idx] = hiprand(&state) % (width - patchSize);  // x coordinate
        coords[2 * idx + 1] = hiprand(&state) % (height - patchSize);  // y coordinate
    }
}

// ==================== Extract 1 Patch ====================

__global__ void extractPatchKernelNaivePara(const unsigned char* image, unsigned char* patch, int startX, int startY, int imageWidth, int patchSize, int numChannels) {
    
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < patchSize && y < patchSize) {
        int imageIndex = ((startY + y) * imageWidth + (startX + x)) * numChannels;
        int patchIndex = (y * patchSize + x) * numChannels;
        for (int channel = 0; channel < numChannels; channel++) {
            patch[patchIndex + channel] = image[imageIndex + channel];
        }
    }
}

// ==================== extractPatches ====================

void extractPatchesNaivePara(const unsigned char* inputImage, const unsigned char* outputImage, int imageWidth, int imageHeight, int patchSize, int numChannels, int numNegativePatches, unsigned char* refPatch, unsigned char* posPatch, unsigned char* negPatches) {

    int numPatches = numNegativePatches + 2;

    // Allocate memory for coordinates on the host and device
    int* hostCoords = (int*)malloc(numPatches * 2 * sizeof(int));
    int* devCoords;
    CHECK_CUDA_ERROR(hipMalloc(&devCoords, numPatches * 2 * sizeof(int)));

    // Generate coordinates sequentially
    for (int i = 0; i < numPatches; i++) {
        generateRandomCoordinatesNaivePara<<<1, 1>>>(devCoords + i * 2, imageWidth, imageHeight, patchSize, 1, time(NULL) + i);
        CHECK_CUDA_ERROR(hipMemcpy(&hostCoords[i * 2], devCoords + i * 2, 2 * sizeof(int), hipMemcpyDeviceToHost));
    }

    // Allocate memory for images on device
    unsigned char *devInputImage, *devOutputImage;
    size_t imageSize = (size_t)imageWidth * imageHeight * numChannels;
    CHECK_CUDA_ERROR(hipMalloc(&devInputImage, imageSize));
    CHECK_CUDA_ERROR(hipMalloc(&devOutputImage, imageSize));
    CHECK_CUDA_ERROR(hipMemcpy(devInputImage, inputImage, imageSize, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(devOutputImage, outputImage, imageSize, hipMemcpyHostToDevice));

    // Allocate memory for patches on device
    unsigned char *devRefPatch, *devPosPatch, *devNegPatches;
    size_t patchSizeBytes = patchSize * patchSize * numChannels * sizeof(unsigned char);
    CHECK_CUDA_ERROR(hipMalloc(&devRefPatch, patchSizeBytes));
    CHECK_CUDA_ERROR(hipMalloc(&devPosPatch, patchSizeBytes));
    CHECK_CUDA_ERROR(hipMalloc(&devNegPatches, numNegativePatches * patchSizeBytes));

    // Allocate streams
    hipStream_t* streams = (hipStream_t*)malloc(numPatches * sizeof(hipStream_t));
    if (streams == NULL) {
        // Error handling
        fprintf(stderr, "Failed to allocate memory for streams\n");
        exit(EXIT_FAILURE);
    }

    // Create streams
    for (int i = 0; i < numPatches; i++) {
        CHECK_CUDA_ERROR(hipStreamCreate(&streams[i]));
    }

    // Configure dimensions for kernel launch
    dim3 blockDims(16, 16);
    dim3 gridDims((patchSize + blockDims.x - 1) / blockDims.x, (patchSize + blockDims.y - 1) / blockDims.y);

    // Extract reference patch
    extractPatchKernelNaivePara<<<gridDims, blockDims, 0, streams[0]>>>(devInputImage, devRefPatch, hostCoords[0], hostCoords[1], imageWidth, patchSize, numChannels);
    CHECK_CUDA_ERROR(hipGetLastError());

    // Extract positive patch
    extractPatchKernelNaivePara<<<gridDims, blockDims, 0, streams[1]>>>(devOutputImage, devPosPatch, hostCoords[0], hostCoords[1], imageWidth, patchSize, numChannels);
    CHECK_CUDA_ERROR(hipGetLastError());

    // Extract negative patches
    for (int i = 0; i < numNegativePatches; i++) {
        //printf("+1");
        int offset = 2 * (i + 1);  // Offset for coordinates
        extractPatchKernelNaivePara<<<gridDims, blockDims, 0, streams[i + 2]>>>(devOutputImage, devNegPatches + i * patchSizeBytes,
                                                                    hostCoords[offset], hostCoords[offset + 1], imageWidth, patchSize, numChannels);
        CHECK_CUDA_ERROR(hipGetLastError());
    }

    // Synchronize all streams
    for (int i = 0; i < numPatches; i++) {
        CHECK_CUDA_ERROR(hipStreamSynchronize(streams[i]));
    }

    // Copy patches to host memory
    CHECK_CUDA_ERROR(hipMemcpy(refPatch, devRefPatch, patchSizeBytes, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(posPatch, devPosPatch, patchSizeBytes, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(negPatches, devNegPatches, numNegativePatches * patchSizeBytes, hipMemcpyDeviceToHost));

    // Destroy streams
    for (int i = 0; i < numPatches; i++) {
        CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
    }

    // Free device memory
    CHECK_CUDA_ERROR(hipFree(devInputImage));
    CHECK_CUDA_ERROR(hipFree(devOutputImage));
    CHECK_CUDA_ERROR(hipFree(devCoords));
    CHECK_CUDA_ERROR(hipFree(devRefPatch));
    CHECK_CUDA_ERROR(hipFree(devPosPatch));
    CHECK_CUDA_ERROR(hipFree(devNegPatches));

    // Free the allocated host memory and stream memory
    free(hostCoords);
    free(streams);
}